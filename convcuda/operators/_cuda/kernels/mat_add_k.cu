
#include <hip/hip_runtime.h>
__global__ void mat_add(float *a, float *b, float *c,
                        int size_x, int size_y) {
    const int i = threadIdx.x,
              j = threadIdx.y,
              real_pos = i * size_y + j;

    if (i < size_x && j < size_y)
        c[real_pos] = a[real_pos] + b[real_pos];
}

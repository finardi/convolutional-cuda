#include "hip/hip_runtime.h"
__global__ void
mat_dot(float *a, float *b, float *c,
        int a_size_x, int a_size_y, int b_size_x, int b_size_y)
{
    const int i = %(N_THREADS_0)s * blockIdx.x + threadIdx.x,
              j = %(N_THREADS_1)s * blockIdx.y + threadIdx.y;

    if (i < a_size_x && j < b_size_y)
    {
        float c_at_ij = 0;
        for (int k = 0; k < a_size_y; k++)
            c_at_ij += a[i * a_size_y + k] * b[k * b_size_y + j];
        c[i * b_size_y + j] = c_at_ij;
    }
}
